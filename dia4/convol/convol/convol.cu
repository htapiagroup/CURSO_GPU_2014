#include "hip/hip_runtime.h"
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define STOP 0
#define START 1

#define BLOCK_X 16
#define BLOCK_Y 16

extern "C" void chrono (int kind, float *time);

__global__ void kconvol (float *gpu_a, float *gpu_b, int pitch, int n) {
  int i, j, l;
  // UP TO YOU : evaluate the coordinate i of the current thread,
  // using blockDim, blockIdx and threadIdx.

  j = blockDim.y*blockIdx.y+threadIdx.y;
  // UP TO YOU : evaluate global 1D index of thread (i,j)
  // in array gpu_a, as a linear combination of i and j.
  // You will have to use the variable 'pitch'

  if ((i >= n) || (j >= n)) return;
  if ((i == 0) || (j == 0) || (i == n-1) || (j == n-1))  {
    gpu_b[l] = gpu_a[l]; // edges are untouched
  }
  else
    // UP TO YOU : fill up the MISSING indices below
    gpu_b[l]=(1.f/5.f)*(                 gpu_a[l-pitch]+\
			gpu_a[/*MISSING*/]  +gpu_a[l]         +gpu_a[l+1]+ \
			                 gpu_a[/*MISSING*/]                 );
}

extern "C" void gpu_convol (float *a, float *b, int n) {
  float *gpu_a;
  float *gpu_b;
  hipError_t err;
  size_t pitch;
  float time;
  err = hipMallocPitch (&gpu_a, &pitch, n*sizeof(float), n);
  if (err != 0) {
    printf ("Error allocating gpu_a: %s\n", hipGetErrorString (err));
    exit (1);
  }
  err = hipMallocPitch (&gpu_b, &pitch, n*sizeof(float), n);
  if (err != 0) {
    printf ("Error allocating gpu_b: %s\n", hipGetErrorString (err));
    exit (1);
  }
  dim3 block (BLOCK_X, BLOCK_Y);
  dim3 grid;
  // UP TO YOU : how many blocks to we want in x (grid.x) and in y (grid.y) ?
  grid.x = ;
  grid.y = ;
  hipMemcpy2D (gpu_a, pitch, a, n*sizeof(float), n*sizeof(float), n, hipMemcpyHostToDevice);
  // NOTE : the chronometer below does not contemplate overhead of memory allocation and
  // memory transfer.
  chrono (START, &time);
  // UP TO YOU : the third variable is missing below in the kernel invocation
  // It is a pitch. Beware ! Does it represent a number of bytes, or a number of floats ?
  kconvol <<<grid, block>>> (gpu_a, gpu_b, /* MISSING */, n);
  err=hipDeviceSynchronize ();
  chrono (STOP, &time);
  printf ("Convolution took  %f sec. on GPU\n", time);
  hipMemcpy2D (b, n*sizeof(float), gpu_b, pitch, n*sizeof(float), n, hipMemcpyDeviceToHost);
  if (err != 0) {
    printf ("%s\n", hipGetErrorString (err));
    exit (1);
  }
  hipFree (gpu_a);
  hipFree (gpu_b);
}
