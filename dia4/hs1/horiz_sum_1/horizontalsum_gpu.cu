#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cudpp.h>
#include <unistd.h>

size_t pitch;


extern "C" double* matrix_alloc_gpu (int size_x, int size_y) {
  void *p;
  hipMallocPitch (&p, &pitch, sizeof(double)*size_x, size_y);
  if (p == NULL) {
    fprintf (stderr, "I cannot reserve device memory for the matrix.\n");
    exit (EXIT_FAILURE);
  }
  return (double *)p;
}

extern "C" void H2D (double *m_gpu, double *m_cpu, int size_x, int size_y) {
  hipMemcpy2D (m_gpu, pitch, m_cpu, size_x*sizeof(double),\
		size_x*sizeof(double), size_y,hipMemcpyHostToDevice);
}

extern "C" void D2H (double *m_cpu, double *m_gpu, int size_x, int size_y) {
  hipMemcpy2D(m_cpu
}


extern "C" void main_gpu (double *d_odata, double *d_idata, int size_x, int size_y) {
  CUDPPHandle theCudpp;
  CUDPPConfiguration config;
  CUDPPHandle scanplan = 0;
  CUDPPResult res;
  
  cudppCreate (&theCudpp);
  config.op = CUDPP_ADD;
  config.datatype = CUDPP_DOUBLE;
  config.algorithm = CUDPP_SCAN;
  config.options = CUDPP_OPTION_BACKWARD | CUDPP_OPTION_EXCLUSIVE;
  //AQUI: Fijese en el tipo de scan efectuado en el host, y complete las opciones (config.options)
  
  //AQUI: fabrique el plan de la reduccion. Cuidado con el valor de 'rowPitch'
  res = cudppPlan (theCudpp, &scanplan, config, size_x,	size_y,	pitch/sizeof(double));
  if (CUDPP_SUCCESS != res) {
    printf("Error creating CUDPPPlan\n");
    exit(-1);
  }
  
  res = cudppMultiScan(scanplan, d_odata, d_idata, size_x*size_y);
  //AQUI: ponga la instruccion para la ejecucion del plan

  if (CUDPP_SUCCESS != res) {
    printf("Error in cudppScan()\n");
    exit(-1);
  }
}
