
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define TILE 16

__global__ void transposeNoBankConflicts(float *odata, float *idata,\
					 int in_pitch, int out_pitch) {
  __shared__ float tile[TILE][TILE+1];
  
  int xIndex = blockIdx.x * TILE + threadIdx.x;
  int yIndex = blockIdx.y * TILE + threadIdx.y;
  int index_in = xIndex + (yIndex)*in_pitch;

  // AQUI: Decomentar o bien la parte 1, o bien la parte 2
  
  //  PARTE 1
  //  xIndex = blockIdx.y * TILE + threadIdx.x;
  //  yIndex = blockIdx.x * TILE + threadIdx.y;
  //////////
  
  // PARTE 2
  //  xIndex = blockIdx.y * TILE + threadIdx.y;
  //  yIndex = blockIdx.x * TILE + threadIdx.x;
  //////////

  int index_out = xIndex + (yIndex)*out_pitch;
  
  tile[threadIdx.y][threadIdx.x] = idata[index_in];
  
  __syncthreads();

  // AQUI: Decomentar o bien la parte 1, o bien la parte 2
  // OJO: la parte que se tiene de decomentar depende de su eleccion mas arriba.
  // Cual combinacion es preferible ?

  // PARTE 1
  //  odata[index_out] = tile[threadIdx.x][threadIdx.y];
  //////

  // PARTE 2
  //  odata[index_out] = tile[threadIdx.y][threadIdx.x];
  //////
}

extern "C" void transpose (float *matrix_in, float *matrix_out, int inp, int outp) {
  dim3 grid, threads;

  if (inp % TILE != 0) {
    fprintf (stderr, "Size problem...\n");
    exit (EXIT_FAILURE);
  }
  if (outp % TILE != 0) {
    fprintf (stderr, "Size problem...\n");
    exit (EXIT_FAILURE);
  }
  grid.x = inp/TILE;
  grid.y = outp/TILE;
  threads.x = TILE;
  threads.y = TILE;
  transposeNoBankConflicts<<<grid,threads>>>(matrix_out,matrix_in,inp,outp);
}
