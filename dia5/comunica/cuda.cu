/**** Archivo: cuda.cu **/
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
__global__ void kernel(int dato,int rank, int *gpu_dato){
	*gpu_dato= dato + rank;
}
extern "C"
int run_kernel(int dato, int rank)
{
int *gpu_dato;
hipMalloc( (void**)&gpu_dato, sizeof(int));
kernel<<<1,1>>>(dato, rank, gpu_dato);
hipMemcpy(&dato, gpu_dato, sizeof(int), hipMemcpyDeviceToHost); 
return dato;
}