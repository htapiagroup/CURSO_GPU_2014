#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
 
/* Size of a block */
#define BLOCK_X 32
#define BLOCK_Y 16

__global__ void kernadd (float* mout, float* min1, float *min2, int nx, int ny, size_t pitch)
{
  int i, j, index;
  /* UP TO YOU edit line below so that the index is correctly evaluated */
  i = blockDim.x * blockIdx.x +threadIdx.x;
  j = blockDim.y * blockIdx.y +threadIdx.y;  

  index= i + j * pitch/sizeof(float);
  
  if ((i < nx) && (j < ny))
    mout[index] = min1[index] + min2[index];
}

/* extern "C" below is because this file follows C++ linking conventions */
/* whereas the companion C file (addition.c) follows C linking conventions */
/* which are different */

extern "C" void AddOnGpu(float* mat_out, float *mat_in1,  float *mat_in2,\
                         int nx, int ny) {

  size_t pitch;			/* Same pitch for all matrices, since
				   they all have same size */

  /* Matrix allocation on device */
  float *mat_out_gpu, *mat_in1_gpu, *mat_in2_gpu;
  /* UP TO YOU : do the allocation below, using hipMallocPitch ()*/


  hipMallocPitch(&mat_out_gpu, &pitch, sizeof(float)*nx, ny);
  hipMallocPitch(&mat_in1_gpu, &pitch, sizeof(float)*nx, ny);
  hipMallocPitch(&mat_in2_gpu, &pitch, sizeof(float)*nx, ny);
  /* The arguments mat_in1 and mat_in2 passed above are on the host. */
  /* UP TO YOU : write below the instructions to copy it to the device */
  /* You'll need to google the function hipMemcpy2D () */
  
  hipMemcpy2D(mat_in1_gpu, pitch, mat_in1, nx*sizeof(float), nx*sizeof(float), ny, hipMemcpyHostToDevice);
  hipMemcpy2D(mat_in2_gpu, pitch, mat_in2, nx*sizeof(float), nx*sizeof(float), ny, hipMemcpyHostToDevice);
  

  /* Grid topology below */
  /* A block is BLOCK_X threads wide by BLOCK_Y threads high */
  dim3 block (BLOCK_X, BLOCK_Y);

  /* UP TO YOU : complete the number of blocks below */

  int n1 = (nx+BLOCK_X-1/BLOCK_X);
  int n2 = (nx+BLOCK_Y-1/BLOCK_Y);

  dim3 grid (n1,n2);

  /* UP TO YOU : kernel invocation */


  kernadd <<< grid, block >>> (mat_out_gpu,mat_in1_gpu, mat_in2_gpu, nx, ny, pitch);
   
  hipDeviceSynchronize();

  /* We now transfer back the matrix from the device to the host */
  /* UP TO YOU : write hipMemcpy2D () instruction below */

  hipMemcpy2D(mat_out, nx * sizeof(float), mat_out_gpu, pitch, nx * sizeof(float),ny,hipMemcpyDeviceToHost);


  /* free memory */
  hipFree(mat_out_gpu);
  hipFree(mat_in1_gpu);
  hipFree(mat_in2_gpu);
}
