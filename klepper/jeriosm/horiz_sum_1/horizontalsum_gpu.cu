#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cudpp.h>
#include <unistd.h>

size_t pitch;


extern "C" double* matrix_alloc_gpu (int size_x, int size_y) {
  void *p;
  hipMallocPitch (&p, &pitch, sizeof(double)*size_x, size_y);
  if (p == NULL) {
    fprintf (stderr, "I cannot reserve device memory for the matrix.\n");
    exit (EXIT_FAILURE);
  }
  return (double *)p;
}

extern "C" void H2D (double *m_gpu, double *m_cpu, int size_x, int size_y) {
  hipMemcpy2D (m_gpu, pitch, m_cpu, size_x*sizeof(double),\
		size_x*sizeof(double), size_y,hipMemcpyHostToDevice);
}

extern "C" void D2H (double *m_cpu, double *m_gpu, int size_x, int size_y) {
  //AQUI: ponga la instruccion correcta de copia de la memoria del
  //device a la memoria del host
}


extern "C" void main_gpu (double *d_odata, double *d_idata, int size_x, int size_y) {
  CUDPPHandle theCudpp;
  CUDPPConfiguration config;
  CUDPPHandle scanplan = 0;
  CUDPPResult res;
  
  cudppCreate (&theCudpp);
  config.op = CUDPP_ADD;
  config.datatype = CUDPP_DOUBLE;
  config.algorithm = CUDPP_SCAN;
  //AQUI: Fijese en el tipo de scan efectuado en el host, y complete las opciones (config.options)
  
  //AQUI: fabrique el plan de la reduccion. Cuidado con el valor de 'rowPitch'
  
  if (CUDPP_SUCCESS != res) {
    printf("Error creating CUDPPPlan\n");
    exit(-1);
  }
  
  //AQUI: ponga la instruccion para la ejecucion del plan

  if (CUDPP_SUCCESS != res) {
    printf("Error in cudppScan()\n");
    exit(-1);
  }
}
