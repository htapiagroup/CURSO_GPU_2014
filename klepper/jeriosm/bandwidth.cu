#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#define STOP 0
#define START 1

/* Play with the following two values */
#define NB 1000000L //Size of array (long integer)
#define MANY 200L    //Number of transfers

/* (over-)Simple chronometer function */
void chrono (int kind, float *time) {
  static clock_t counts;
  if (kind == START) {
    *time = 0.0;
    counts = clock();
    return;
  }
  if (kind == STOP) {
    *time = ((float)(clock()-counts))/((float)CLOCKS_PER_SEC);
  }
}

int main () {
  float *ptr, *gpu_ptr;
  hipError_t err;
  float time, number_of_Gbytes;
  long i, j;
  
  /* Dynamic allocations below */
  //  Allocate ptr on host below
    ptr = (float *)malloc(NB * sizeof(float));
  // Allocate gpu_ptr on device below
 err = hipMalloc(&gpu_ptr,sizeof(float)*NB);

  /* Some error handling */
  if (ptr == NULL) {
    printf ("Not enough memory on host\n");
    exit (1);
  }
  if (err != 0) {
    printf  ("Pb allocating memory on device. Reason:\n");
    printf ("%s\n", hipGetErrorString (err));
    exit (1);
  }

  /* Are the following two lines necessary ? */
  for (i = 0; i < NB; i++) 
    ptr[i] = 9.0;

  /* Transfer loop below */
  chrono (START, &time);
  for (j = 0; j < MANY; j++) {
    // perform memory transfer here
//    cudaMemcpy(gpu_ptr,ptr,sizeof(float)*NB,cudaMemcpyHostToDevice);
    hipMemcpy(gpu_ptr,ptr,sizeof(float)*NB,hipMemcpyDeviceToHost);
  }
  chrono (STOP, &time);

  /* Output results */
  number_of_Gbytes = (MANY*(float)sizeof(float)*NB/1024./1024./1024.);
  printf ("%f Gbytes transfered in %f seconds\n", number_of_Gbytes, time);
  printf ("Bandwidth = %f Gb/s\n", number_of_Gbytes / time);

  /* Clean the place */
  hipFree (gpu_ptr);
  free (ptr);
  return 0;
}
  
