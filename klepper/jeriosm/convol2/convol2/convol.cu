#include "hip/hip_runtime.h"
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define STOP 0
#define START 1

#define BLOCK_X 16
#define BLOCK_Y 16

extern "C" void chrono (int kind, float *time);

__global__ void kconvol (float *gpu_a, float *gpu_b, int pitch, int n) {
  int ig, jg, lg, il, jl, ll;
  __shared__ float la[(BLOCK_X+2)*(BLOCK_Y+2)];
  __shared__ float lb[(BLOCK_X+2)*(BLOCK_Y+2)];
 
 // A thread now has two sets of coordinates : 
  // (ig, jg) in the global array
  // (il, jl) in the local array (shared) of size (BLOCK_X+2)*(BLOCK_Y+2)
  ig = blockDim.x*blockIdx.x+threadIdx.x;
  jg = blockDim.y*blockIdx.y+threadIdx.y;
  lg = ig+jg*pitch;
  // UP TO YOU : write below the indices il and jl
  il = threadIdx.x+1;
  jl = threadIdx.y+1;
  ll = il+jl*(BLOCK_X+2);
 
 // What does the following line correspond to ?
  la[ll] = gpu_a[lg];
  if ((il == 1) && (ig > 0)) // What does the following line correespond to ?
    la[ll-1] = gpu_a[lg-1];
  if ((jl == 1) && (jg > 0))
    la[ll-BLOCK_X-2] = gpu_a[lg-pitch];
  if ((il == BLOCK_X) && (ig < n-1)) // UP TO YOU The following line is missing. Find out what was intended
    la[ll-BLOCK_X-2]= gpu_a[lg+pitch];
  if ((jl == BLOCK_Y) && (jg < n-1)) // UP TO YOU Find out the missing offset of local array below
    la[ll+BLOCK_X+2] = gpu_a[lg+pitch];
  __syncthreads ();
  if ((ig >= n) || (jg >= n)) return;
  if ((ig == 0) || (jg == 0) || (ig == n-1) || (jg == n-1))  {
    lb[ll] = la[ll];
  }
  else
    /* UP TO YOU : fill up below the missing indices */
    lb[ll]=(1.f/5.f)*(                +la[ll-BLOCK_X-2]+	     \
		      la[ll-1]        +la[ll]             +la[ll+1]+ \
		                      +la[ll+BLOCK_X+2]);
if((ig<n) &&(ig>0))  
gpu_b[lg] = lb[ll];
}

extern "C" void gpu_convol (float *a, float *b, int n) {
  float *gpu_a;
  float *gpu_b;
  hipError_t err;
  size_t pitch;
  float time;
  err = hipMallocPitch (&gpu_a, &pitch, n*sizeof(float), n);
  if (err != 0) {
    printf ("Error allocating gpu_a: %s\n", hipGetErrorString (err));
    exit (1);
  }
  err = hipMallocPitch (&gpu_b, &pitch, n*sizeof(float), n);
  if (err != 0) {
    printf ("Error allocating gpu_b: %s\n", hipGetErrorString (err));
    exit (1);
  }
  dim3 block (BLOCK_X, BLOCK_Y);
  dim3 grid;
  grid.x = (n-1)/BLOCK_X+1;
  grid.y = (n-1)/BLOCK_Y+1;
  hipMemcpy2D (gpu_a, pitch, a, n*sizeof(float), n*sizeof(float), n, hipMemcpyHostToDevice);
  chrono (START, &time);
  kconvol <<<grid, block>>> (gpu_a, gpu_b, pitch/sizeof(float), n);
  err=hipDeviceSynchronize ();
  chrono (STOP, &time);
  printf ("Convolution took  %f sec. on GPU\n", time);
  hipMemcpy2D (b, n*sizeof(float), gpu_b, pitch, n*sizeof(float), n, hipMemcpyDeviceToHost);
  
  if (err != 0) {
    printf ("%s\n", hipGetErrorString (err));
    exit (1);
  }
  hipFree (gpu_a);
  hipFree (gpu_b);
}
