#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKSIZE 256

__global__ void kern_set_val (float *gpu_ptr, float value, int nb) {
  int i;
  i= 2*(blockDim.x * blockIdx.x+2*threadIdx.x);
  gpu_ptr[i] = value;
}

extern "C" void set_value (float *ptr, float value, int nb) {
  float *gpu_ptr;
  hipMalloc (&gpu_ptr, sizeof(float)*nb);

  //UP TO YOU : write kernel invocation here
  kern_set_val <<<(nb/2+BLOCKSIZE-1/BLOCKSIZE),BLOCKSIZE>>>(gpu_ptr,value,nb);

  hipDeviceSynchronize ();

  hipMemcpy(ptr,gpu_ptr,nb*sizeof(float),hipMemcpyDeviceToHost);

  hipFree (gpu_ptr);
}
