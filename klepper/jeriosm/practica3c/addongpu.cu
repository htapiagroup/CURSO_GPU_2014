#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <time.h>
/* Size of a block */
#define BLOCK_X 32
#define BLOCK_Y 16
#define MANY 200
#define STOP 0
#define START 1

__global__ void kernadd (float* mout, float* min1, float *min2, int nx, int ny, size_t pitch)
{
  int i, j, index;
  /* UP TO YOU edit line below so that the index is correctly evaluated */
  i = blockDim.x * blockIdx.x + threadIdx.x;
  j = blockDim.y * blockIdx.y + threadIdx.y;
  index = i + j * pitch / sizeof(float);
  if ((i < nx) && (j < ny))
    mout[index] = min1[index] + min2[index];
}

/* extern "C" below is because this file follows C++ linking conventions */
/* whereas the companion C file (addition.c) follows C linking conventions */
/* which are different */

extern "C" void chrono (int kind, float *time);

extern "C" float AddOnGpu(float* mat_out, float *mat_in1,  float *mat_in2,int nx, int ny) {

  size_t pitch;			/* Same pitch for all matrices, since
				   they all have same size */

  /* Matrix allocation on device */
  float *mat_out_gpu, *mat_in1_gpu, *mat_in2_gpu;
  /* UP TO YOU : do the allocation below, using hipMallocPitch ()*/
  hipMallocPitch (&mat_out_gpu, &pitch, sizeof(float) * nx, ny);
  hipMallocPitch (&mat_in1_gpu, &pitch, sizeof(float) * nx, ny);
  hipMallocPitch (&mat_in2_gpu, &pitch, sizeof(float) * nx, ny);
  /* The arguments mat_in1 and mat_in2 passed above are on the host. */
  /* UP TO YOU : write below the instructions to copy it to the device */
  /* You'll need to google the function hipMemcpy2D () */
  hipMemcpy2D(mat_in1_gpu,pitch,mat_in1,nx*sizeof(float),nx*sizeof(float),ny,hipMemcpyDeviceToHost);
  hipMemcpy2D(mat_in2_gpu,pitch,mat_in2,nx*sizeof(float),nx*sizeof(float),ny,hipMemcpyDeviceToHost);
  /* Grid topology below */
  /* A block is BLOCK_X threads wide by BLOCK_Y threads high */
  dim3 block (BLOCK_X, BLOCK_Y);

  /* UP TO YOU : complete the number of blocks below */

  int n1 = (nx+BLOCK_X-1)/BLOCK_X;
  int n2 = (nx+BLOCK_Y-1)/BLOCK_Y;;

  dim3 grid (n1,n2);
  
  int count;
  float time;
  chrono (START, &time);
  /* UP TO YOU : kernel invocation */
  for (count = 0;count < MANY; count++){
    kernadd <<< grid,block >>> (mat_out_gpu, mat_in1_gpu,mat_in2_gpu,nx,ny,pitch);
    hipDeviceSynchronize();
  }
  chrono (STOP, &time);

  /* We now transfer back the matrix from the device to the host */
  /* UP TO YOU : write hipMemcpy2D () instruction below */
  
  hipMemcpy2D (mat_out,sizeof(float)*nx,mat_out_gpu,pitch,nx*sizeof(float),ny,hipMemcpyDeviceToHost);

  /* free memory */
  hipFree(mat_out_gpu);
  hipFree(mat_in1_gpu);
  hipFree(mat_in2_gpu);
  return time/float(MANY);
}
